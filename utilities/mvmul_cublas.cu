#include "utilities.h"

//matrix-vector multiplication 
//y = α op ( A ) x + β y
void mvmul_cublas(std::vector<float> A, hipblasOperation_t transa,  std::vector<float> x, std::vector<float> &y, int Arows, int Acols ){

   int lda = Arows ;          //leading dimension of A; 
   const float alf = 1;
   const float bet = 0 ;
   const float *alpha = &alf ;
   const float *beta = &bet ;

   //declare gpu device matrix 
   thrust::device_vector<float> d_A(Arows* Acols), d_x(x.size()), d_y(Acols);

   //convert vector to cublas data type 
   thrust::copy(A.begin(),A.end(),d_A.begin());
   thrust::copy(x.begin(),x.end(),d_x.begin());

   // Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);

   // Do the actual multiplication
   hipblasSgemv(handle,
         transa,
         Arows,
         Acols,
         alpha,
         thrust::raw_pointer_cast(&d_A[0]),
         lda,
         thrust::raw_pointer_cast(&d_x[0]),
         1,		//incx
         beta,
         thrust::raw_pointer_cast(&d_y[0]),
         1 ); //incry

   //Copy data back to cpu 
   thrust::copy(d_y.begin(),d_y.end(),y.begin());


   // Destroy the handle
   hipblasDestroy(handle);

}

