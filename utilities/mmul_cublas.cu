#include "utilities.h"

// C = alpha*op(A)*op(B) + beta*C ; 
void mmul_cublas(std::vector<float> A, hipblasOperation_t transa,  std::vector<float> B, hipblasOperation_t transb ,std::vector<float> &C, int Arows, int Acols, int Bcols){

   int Brows = Acols ;
   int Crows = Arows;
   int Ccols = Bcols ;

   int lda = Arows ;          //leading dimension of A; 
   int ldb = Brows ;          //leading dimenstion of B; 
   int ldc = Crows ;
   const float alf = 1;
   const float bet = 0 ;
   const float *alpha = &alf ;
   const float *beta = &bet ;

   //declare gpu device matrix 
   thrust::device_vector<float> d_A(Arows* Acols), d_B(Brows*Bcols), d_C(Crows*Ccols);

   //convert vector to cublas data type 
   thrust::copy(A.begin(),A.end(),d_A.begin());
   thrust::copy(B.begin(),B.end(),d_B.begin());

   // Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);

   // Do the actual multiplication
   hipblasSgemm(handle,
         transa,
         transb,
         Arows,
         Bcols,
         Acols,
         alpha,
         thrust::raw_pointer_cast(&d_A[0]),
         lda,
         thrust::raw_pointer_cast(&d_B[0]),
         ldb,
         beta,
         thrust::raw_pointer_cast(&d_C[0]),
         ldc);

   //Copy data back to cpu 
   thrust::copy(d_C.begin(),d_C.end(),C.begin());


   // Destroy the handle
   hipblasDestroy(handle);


}

